#include <iostream>
#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

 #define ALIVE 1
 #define DEAD 0
 #define N WX*WY
 #define BLOCK_SIZE 16
 
 // Read Table from file
 void read_from_file(int *X, char *filename, int WX, int WY){
  FILE *fp = fopen(filename, "r+");
  int size = fread(X, sizeof(int), WX*WY, fp);
  printf("elements: %d\n", size);  
  fclose(fp);
} 
 
 // Save Table in file
 void save_table(int *X, int WX, int WY){
  FILE *fp;
  char filename[20];
  sprintf(filename, "cuda_gm_table%dx%d.bin", WX, WY);
  printf("Saving table in file %s\n\n", filename);
  fp = fopen(filename, "w+");
  fwrite(X, sizeof(int), WX*WY, fp);
  fclose(fp);
}
 
 // Device function: get world array index from world coordinates
 __device__ int getId(int x, int y, int WX, int WY)
 {
 // cyclic boundary conditions:
 while(x >= WX)
 x -= WX;

 while(x < 0)
 x += WX;

 while(y >= WY)
 y -= WY;

 while(y < 0)
 y += WY;

 return x + y * WX;
 }

 // Kernel:
 __global__ void runConway(int *world, int *sites, int WX, int WY)
 {
 // get the world coordinate:
 int x = blockIdx.x * blockDim.x + threadIdx.x;
 int y = blockIdx.y * blockDim.y + threadIdx.y;

 // id in 1D array which represents the world:
 int id = getId(x, y, WX, WY);

  // determine new state by rules of Conway’s Game of Life:
 int state = world[id];
 
 // calculate number of alive neighbors:
 int aliveNeighbors = 0;

 for(int x_offset=-1; x_offset<=1; x_offset++){
	for(int y_offset=-1; y_offset<=1; y_offset++){
		if(x_offset != 0 || y_offset != 0){ // don’t count itself
			int neighborId = getId(x + x_offset, y + y_offset, WX, WY);
			aliveNeighbors += world[neighborId];
		
		}
	}
 }

 // decide about new state:
 if(state == ALIVE)
 {
 if(aliveNeighbors < 2 || aliveNeighbors > 3)
 sites[id] = DEAD;
 else
 sites[id] = ALIVE;
 }
 else // if DEAD
 {
 if(aliveNeighbors == 3)
 sites[id] = ALIVE;
 }
  __syncthreads();
 }

 // Host function (CPU Code)
 int main (int argc, char * argv[])
 {
  char *filename = argv[1];
  int WX= atoi(argv[2]);
  int WY= atoi(argv[2]); 
  hipEvent_t start, stop;
  float elapsedTime;
  
  printf("Reading %dx%d table from file %s\n", WX, WY, filename);
  int *world = (int *)malloc(WX*WY*sizeof(int));  
  int *tempsites= (int *)malloc(WX*WY*sizeof(int));  
  
  read_from_file(world, filename, WX, WY);
 
 // input results: 10x10 middle part of the world for easy checking
  printf("---->FIRST WORLD<---- 10x10 middle part\n");
 for(int y=WY/2; y<(WY/2+10); y++){
	for(int x=WX/2; x<(WX/2+10); x++){
		std::cout<<world[x + WX*y];
	}
	std::cout<<"\n";
 }
 std::cout<<"\n";
 
  // number of steps in Conway’s Game of Life:
 int iterations = atoi(argv[3]);


 // allocate memory on CUDA device:
 int *pDevWorld; // pointer to the data on the CUDA Device
 int *pTempDevWorld; // pointer to the data on the CUDA Device
 hipMalloc((int**)&pDevWorld, N*sizeof(int));
 hipMalloc((int**)&pTempDevWorld, N*sizeof(int));

 // copy data to CUDA device:
 hipMemcpy(pDevWorld, world, N*sizeof(int), hipMemcpyHostToDevice);
 hipMemcpy(pTempDevWorld, world, N*sizeof(int), hipMemcpyHostToDevice);
  
 // set block and grid dimensions:
 dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE, 1);
 int GridX = (int)ceil(WX/(float)BLOCK_SIZE);
 int GridY = (int)ceil(WX/(float)BLOCK_SIZE);
 dim3 gridSize(GridX, GridY, 1); 
 
 printf("Blocks= %d\n", GridX*GridY);
 printf("Threads per Block= %d\n", BLOCK_SIZE*BLOCK_SIZE);
 printf("Total Threads=Blocks*Threads per Block= %d threads\n\n", BLOCK_SIZE*BLOCK_SIZE*GridX*GridY);
 // get time of start
 hipEventCreate(&start);
 hipEventRecord(start,0);
 
  // run the defined number of steps:
 for(int i=0; i<iterations; i++){
 // execute kernel function on GPU:
 runConway<<<gridSize, blockSize>>>(pDevWorld, pTempDevWorld, WX, WY);

  // Swap our grids and iterate again
        tempsites = pDevWorld;
        pDevWorld = pTempDevWorld;
        pTempDevWorld = tempsites;
 }
 
 // get time of end
 hipEventCreate(&stop);
 hipEventRecord(stop,0);
 hipEventSynchronize(stop);
 
 // copy data back from CUDA Device to ’data’ array:
 hipMemcpy(world, pDevWorld, N*sizeof(int), hipMemcpyDeviceToHost);

 // free memory on the CUDA Device:
 hipFree(pDevWorld);
 hipFree(pTempDevWorld);

 // output results: 10x10 middle part of the world for easy checking
 printf("---->FINAL WORLD 10x10 middle part<----\n"); 
 for(int y=WY/2; y<(WY/2+10); y++){
	for(int x=WX/2; x<(WX/2+10); x++){
		std::cout<<world[x + WX*y];
	}
	std::cout<<"\n";
 }

 std::cout<<"\n";
 
 save_table(world, WX, WY);
 
 hipEventElapsedTime(&elapsedTime, start,stop);
 printf("Elapsed time (Global Memory) : %f ms\n" ,elapsedTime);
 
 free(world);
 return 0;
 }

