#include <iostream>
#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

 #define ALIVE 1
 #define DEAD 0
 #define N WX*WY
 #define BLOCK_SIZE 16
 #define SMEM (BLOCK_SIZE+2)
 
 // Read Table from file
 void read_from_file(int *X, char *filename, int WX, int WY){
  FILE *fp = fopen(filename, "r+");
  int size = fread(X, sizeof(int), WX*WY, fp);
  printf("elements: %d\n", size);  
  fclose(fp);
} 
 
 // Save Table in file
 void save_table(int *X, int WX, int WY){
  FILE *fp;
  char filename[20];
  sprintf(filename, "cuda_sm_table%dx%d.bin", WX, WY);
  printf("Saving table in file %s\n\n", filename);
  fp = fopen(filename, "w+");
  fwrite(X, sizeof(int), WX*WY, fp);
  fclose(fp);
}
 
 // Device function: get world array index from world coordinates
 __device__ int getId(int x, int y, int WX, int WY)
 {
  return x + y * WX;
 }

 // Kernel:
 __global__ void runConway(int *world, int *sites, int WX, int WY )
 {
 // get the world coordinate:
 int x = blockIdx.x * blockDim.x + threadIdx.x;
 int y = blockIdx.y * blockDim.y + threadIdx.y;
 
 int id = getId(x, y, WX, WY);
 
 int i = threadIdx.y;
 int j = threadIdx.x;
   
 
  // Declare the shared memory on a per block level
  __shared__ int sgrid[SMEM*SMEM];
 
 if ((x<WX)&&(y<WY))
	sgrid[SMEM+1+j+SMEM*i] = world[id];
  
  __syncthreads();
  
 __shared__ unsigned int y_min_off,y_max_off,x_min_off,x_max_off;
  
 // load upper line
	if (threadIdx.y == 0)
	{	if (y>0)
			y_min_off = y-1;
		else
			y_min_off = WY-1;

		sgrid[j+1] = world[getId(x,y_min_off,WX,WY)];
    }
	
	__syncthreads();
	
	// load lower line
	if (i ==(BLOCK_SIZE-1))
	{	if (y < WY-1)
			y_max_off = y+1;
		else
			y_max_off = 0;

		sgrid[(SMEM*(SMEM-1)+1)+j] = world[getId(x,y_max_off,WX,WY)];	
	}
	
	__syncthreads();	
	
	// load left line
	if (j ==0)
	{	if (x>0)
			x_min_off = x-1;
		else
			x_min_off = WX-1;

		sgrid[i*SMEM+SMEM] = world[getId(x_min_off,y,WX,WY)];	
	}
	
	__syncthreads();
	
	// load rigth line
	if (j ==(BLOCK_SIZE-1))
	{	if (x < WX -1)
			x_max_off = x+1;
		else
			x_max_off = 0;

		sgrid[i*SMEM+SMEM+SMEM-1] = world[getId(x_max_off,y,WX,WY)];	
	}
		
	__syncthreads();

	if (j == 0 && i == 0)
	{ 	sgrid[0] = world[getId(x_min_off,y_min_off,WX,WY)];	
		sgrid[SMEM-1] = world[getId(x_max_off,y_min_off,WX,WY)];
		sgrid[SMEM*(SMEM-1)] = world[getId(x_min_off,y_max_off,WX,WY)];
		sgrid[SMEM*SMEM-1] = world[getId(x_max_off,y_max_off,WX,WY)];
	}

	__syncthreads();

	if ((x<WX)&&(y<WY)){
  // determine new state by rules of Conway’s Game of Life:
 int state = sgrid[j+1+(i+1)*SMEM];
 
 // calculate number of alive neighbors:
 int aliveNeighbors = 0;

 for(int x_offset=-1; x_offset<=1; x_offset++){
	for(int y_offset=-1; y_offset<=1; y_offset++){
		if(x_offset != 0 || y_offset != 0){ // don’t count itself
			aliveNeighbors += sgrid[j+1+x_offset+(i+1+y_offset)*SMEM];
		
		}
	}
 }

 // decide about new state:
 if(state == ALIVE)
 {
 if(aliveNeighbors < 2 || aliveNeighbors > 3)
 sites[id] = DEAD;
 else
 sites[id] = ALIVE;
 }
 else // if DEAD
 {
 if(aliveNeighbors == 3)
 sites[id] = ALIVE;
 }
  __syncthreads();
 }
 }

 
 // Host function (CPU Code)
 int main (int argc, char * argv[])
 {
  char *filename = argv[1];
  int WX= atoi(argv[2]);
  int WY= atoi(argv[2]); 
  hipEvent_t start, stop;
  float elapsedTime;
    
  printf("Reading %dx%d table from file %s\n", WX, WY, filename);
  int *world = (int *)malloc(WX*WY*sizeof(int));  
  int *tempsites= (int *)malloc(WX*WY*sizeof(int));  
  
  read_from_file(world, filename, WX, WY);
 
 // input results: 10x10 middle part of the world for easy checking
  printf("---->FIRST WORLD<---- 10x10 middle part\n");
 for(int y=WY/2; y<(WY/2+10); y++){
	for(int x=WX/2; x<(WX/2+10); x++){
		std::cout<<world[x + WX*y];
	}
	std::cout<<"\n";
 }
 std::cout<<"\n";
 
  // number of steps in Conway’s Game of Life:
 int iterations = atoi(argv[3]);


 // allocate memory on CUDA device:
 int *pDevWorld; // pointer to the data on the CUDA Device
 int *pTempDevWorld; // pointer to the data on the CUDA Device
 hipMalloc((int**)&pDevWorld, N*sizeof(int));
 hipMalloc((int**)&pTempDevWorld, N*sizeof(int));

 // copy data to CUDA device:
 hipMemcpy(pDevWorld, world, N*sizeof(int), hipMemcpyHostToDevice);
 hipMemcpy(pTempDevWorld, world, N*sizeof(int), hipMemcpyHostToDevice);
  
 // set block and grid dimensions:
 dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE, 1);
 int GridX = (int)ceil(WX/(float)BLOCK_SIZE);
 int GridY = (int)ceil(WX/(float)BLOCK_SIZE);
 dim3 gridSize(GridX,GridY, 1); 
 
 int totalThreadNum=blockSize.x*blockSize.y*gridSize.x*gridSize.y;
 
 printf("Blocks= %d\n", gridSize.x*gridSize.y);
 printf("Threads per Block= %d\n", blockSize.x*blockSize.y);
 printf("Total Threads=Blocks*Threads per Block= %d threads\n\n", totalThreadNum);
 
 
 int dataPerThread  = N / totalThreadNum;
 printf("Data Per Thread= %d\n\n", dataPerThread);
 
 // get time of start
 hipEventCreate(&start);
 hipEventRecord(start,0);
 
  // run the defined number of steps:
 for(int i=0; i<iterations; i++){
 // execute kernel function on GPU:
 runConway<<<gridSize, blockSize>>>(pDevWorld, pTempDevWorld, WX, WY);

  // Swap our grids and iterate again
        tempsites = pDevWorld;
        pDevWorld = pTempDevWorld;
        pTempDevWorld = tempsites;
 }
 
 // get time of end
 hipEventCreate(&stop);
 hipEventRecord(stop,0);
 hipEventSynchronize(stop);
 
 // copy data back from CUDA Device to ’data’ array:
 hipMemcpy(world, pDevWorld, N*sizeof(int), hipMemcpyDeviceToHost);

 // free memory on the CUDA Device:
 hipFree(pDevWorld);
 hipFree(pTempDevWorld);

 // output results: 10x10 middle part of the world for easy checking
 printf("---->FINAL WORLD 10x10 middle part<----\n"); 
 for(int y=WY/2; y<(WY/2+10); y++){
	for(int x=WX/2; x<(WX/2+10); x++){
		std::cout<<world[x + WX*y];
	}
	std::cout<<"\n";
 }

 std::cout<<"\n";
 
 save_table(world, WX, WY);
 
 hipEventElapsedTime(&elapsedTime, start,stop);
 printf("Elapsed time (Shared Memory) : %f ms\n" ,elapsedTime);
 
 free(world);
 return 0;
 }

